#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2018. FastCorrelation
  Computes the correlation of two signals (autocorrelation if the signals are the same).
  
  USAGE:
     $ cat two_column_file  | correlation -N [N]  > corr.dat
     
     The output will have three columns:
	lag corr(lag) std(lag)

     std = sqrt(<corr^2> - <corr>^2)  


    If you have several files with realizations of the same signal, say signal1, signal2,...signalN
      you can interleave them to use with correlation as:
     $ paste -d'\n' signal* | correlation ... > corr.dat

  INPUT FORMAT:
     The input must have two columns with the following order:
       A_t0_signal1 B_t0_signal1
       A_t0_signal2 B_t0_signal2
       ...
       A_t0_signal_nsignals   B_t0_signal_nsignals
       A_t1_signal_nsignals   B_t1_signal_nsignals
       ...
       A_t_N_signal_nsignals  B_t_N_signal_nsignals
       
     
  OPTIONS:
    -N: signal length
    -nsignals: number of signals in the file (they all must have length N) (default is 1).
    -windowSize: If present the signal will be cut in pieces of windowSize size for averaging (default is N).
    -maxLag: Compute up to lag maxLag (default is N). A lower value will increase performance and the results will be numerically identical up to maxLag.
    -noPad: If not present the signall will be padded with zeros up to maxLag in FFT mode. (default is false)
    -scale: Scale mode (default is biased), can be:
          none:   Return the unscaled correlation, R.
          biased: Return the biased average, R/N
	  unbiased: Return the unbiased average, R(t)/(N-t)
    -prec: float or double, specify the precision mode (default is float).

    -h or -help: Print this info.

   
 */



#include"parseArguments.h"
#include"correlationGPU.cuh"

using namespace FastCorrelation;
void print_help();
int main(int argc, char *argv[]){


  //Parse command line options
  if(checkFlag(argc, argv, "-h") or checkFlag(argc, argv, "-help")){print_help(); exit(1);}
  
  int numberElements;   if(!parseArgument(argc, argv, "-N", &numberElements)){print_help();exit(1);}
  int nsignals = 1;     parseArgument(argc, argv, "-nsignals", &nsignals);
  int windowSize = numberElements;    parseArgument(argc, argv, "-windowSize", &windowSize);
  int maxLag = windowSize;   if(parseArgument(argc, argv, "-maxLag", &maxLag)){
    if(maxLag>windowSize){
      std::cerr<<"WARNING!: You should not ask for a lag time larger than the window size!"<<std::endl;
    }
  }
  bool padSignal = !checkFlag(argc, argv, "-noPad");  
  ScaleMode scaleMode = ScaleMode::biased;
  {//Scale mode
    std::string scale="biased"; parseArgument(argc, argv, "-scale", &scale);
    if(scale.compare("biased") == 0)        scaleMode = ScaleMode::biased;
    else if(scale.compare("unbiased") == 0) scaleMode = ScaleMode::unbiased;
    else if(scale.compare("none") == 0)     scaleMode = ScaleMode::none;
    else{ std::cerr<<"ERROR: Unrecognized scale mode!"<<std::endl; print_help(); exit(1);}
  }
  
  //Precision mode
  std::string prec="float"; parseArgument(argc, argv, "-prec", &prec);
  

  //Compute correlation

  
  if(prec.compare("float") == 0){
    correlationGPUFFT<float>(stdin,
			     numberElements,
			     nsignals,
			     windowSize,
			     maxLag,
			     padSignal,
			     scaleMode);
  }
  else if(prec.compare("double") == 0){
    correlationGPUFFT<double>(stdin,
			     numberElements,
			     nsignals,
			     windowSize,
			     maxLag,
			     padSignal,
			     scaleMode);
  }
  else{
    std::cerr<<"ERROR: Unrecognized precision mode!! Select float or double!."<<std::endl;
    print_help();
    exit(1);
  }
 

  return 0;
}

void print_help(){
std::cerr<<"Raul P. Pelaez 2018. FastCorrelation"<<std::endl;
std::cerr<<"  Computes the correlation of two signals (autocorrelation if the signals are the same)."<<std::endl;
std::cerr<<"  "<<std::endl;
std::cerr<<"  USAGE:"<<std::endl;
std::cerr<<"     $ cat two_column_file  | correlation -N [N]  > corr.dat"<<std::endl;
std::cerr<<"     "<<std::endl;
std::cerr<<"     The output will have three columns:"<<std::endl;
std::cerr<<"	lag corr(lag) std(lag)"<<std::endl;
std::cerr<<""<<std::endl;
std::cerr<<"     std = sqrt(<corr^2> - <corr>^2)  "<<std::endl;
std::cerr<<""<<std::endl;
std::cerr<<""<<std::endl;
std::cerr<<"    If you have several files with realizations of the same signal, say signal1, signal2,...signalN"<<std::endl;
std::cerr<<"      you can interleave them to use with correlation as:"<<std::endl;
std::cerr<<"     $ paste -d'\n' signal* | correlation ... > corr.dat"<<std::endl;
std::cerr<<""<<std::endl;
std::cerr<<"  INPUT FORMAT:"<<std::endl;
std::cerr<<"     The input must have two columns with the following order:"<<std::endl;
std::cerr<<"       A_t0_signal1 B_t0_signal1"<<std::endl;
std::cerr<<"       A_t0_signal2 B_t0_signal2"<<std::endl;
std::cerr<<"       ..."<<std::endl;
std::cerr<<"       A_t0_signal_nsignals   B_t0_signal_nsignals"<<std::endl;
std::cerr<<"       A_t1_signal_nsignals   B_t1_signal_nsignals"<<std::endl;
std::cerr<<"       ..."<<std::endl;
std::cerr<<"       A_t_N_signal_nsignals  B_t_N_signal_nsignals"<<std::endl;
std::cerr<<"       "<<std::endl;
std::cerr<<"     "<<std::endl;
std::cerr<<"  OPTIONS:"<<std::endl;
std::cerr<<"    -N: signal length"<<std::endl;
std::cerr<<"    -nsignals: number of signals in the file (they all must have length N) (default is 1)."<<std::endl;
std::cerr<<"    -windowSize: If present the signal will be cut in pieces of windowSize size for averaging (default is N)."<<std::endl;
std::cerr<<"    -maxLag: Compute up to lag maxLag (default is N). A lower value will increase performance and the results will be numerically identical up to maxLag."<<std::endl;
std::cerr<<"    -noPad: If not present the signall will be padded with zeros up to maxLag in FFT mode. (default is false)"<<std::endl;
std::cerr<<"    -scale: Scale mode (default is biased), can be:"<<std::endl;
std::cerr<<"          none:   Return the unscaled correlation, R."<<std::endl;
std::cerr<<"          biased: Return the biased average, R/N"<<std::endl;
std::cerr<<"	  unbiased: Return the unbiased average, R(t)/(N-t)"<<std::endl;
std::cerr<<"    -prec: float or double, specify the precision mode (default is float)."<<std::endl;
std::cerr<<""<<std::endl;
std::cerr<<"    -h or -help: Print this info."<<std::endl;

}
